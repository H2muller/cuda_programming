#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from here


__global__ void reductionPhaseKernel() {
  // XY[22 * BLOCK_SIZE] from shared memory
  for (unsigned int stride = 1; stride <= BLOCK_SIZE; stride *= 2) {
    int localIdx = (threadIdx.x + 1) * stride * 2 - 1;
    if (localIdx < 2 * BLOCK_SIZE) {
      sharedArray[localIdx] += sharedArray[localIdx - stride];
    __syncthreads();
    }
  }
}

__global__ void postReductionReversePhase() {
  for (unsigned int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2) {
    int localIdx = (threadIdx.x + 1) * stride * 2 - 1;
    if (localIdx < 2 * BLOCK_SIZE) {
      XY[localIdx + stride] += XY[localIdx];
    }
  }
  __syncthreads();
  if (i < InputSize) {
    Output[i] = sharedArray[threadIdx.x];
  }
}


__global__ void singlePassScan(float *input, float *output, int len, int loadIdx, int loadStride) {
  
  __shared__ float sharedArray[BLOCK_SIZE * 2];

  if(loadIdx < len){
        sharedArray[threadIdx.x] = input[loadIdx];
  } else {
        sharedArray[threadIdx.x] = 0; 
  }

  if(loadIdx + loadStride < len){
        sharedArray[threadIdx.x + blockDim.x] = input[loadIdx + loadStride];
  } else {
        sharedArray[threadIdx.x + blockDim.x] = 0;
  }
  __syncthreads();

  
  // reduction phase
  for (int stride = 1; stride <= blockDim.x; stride *= 2) {
    __syncthreads();
    int localIdx = (threadIdx.x + 1) * stride * 2 - 1;
    if (localIdx < 2 * blockDim.x) {
      sharedArray[localIdx] += sharedArray[localIdx - stride];
    }
  }

  // reverse phase
  for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
    __syncthreadS();
    int localIdx = (threadIdx.x + 1) * stride * 2 - 1;
    if (localIdx + stride < 2 * blockDim.x) {
      sharedArray[localIdx + stride] += sharedArray[localIdx];
    }
  }

  // store partial results to output
  __syncthreads();
  if (2 * blockIdx.x * blockDim.x + threadIdx.x < len) {
    output[2 * blockIdx.x * blockDim.x + threadIdx.x] = sharedArray[threadIdx.x];
  }
  if (2 * blockIdx.x * blockDim.x + threadIdx.x + blockDim.x < len) {
    output[2 * blockIdx.x * blockDim.x + threadIdx.x + blockDim.x] = sharedArray[threadIdx.x + blockDim.x];
  }
}



void recursiveScan (float *input, float *output, int len) {
  
  // FIRST PASS
  dim3 dimBlock(BLOCK_SIZE, 1, 1);
  dim3 dimGrid(ceil(len / (float)BLOCK_SIZE), 1, 1);

  int firstLoadIdx = 2 * blockIdx.x * blockDim.x + threadIdx.x;
  int firstLoadStride = blockDim.x;

  singlePassScan<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, numElements, firstLoadIdx, firstLoadStride);

  // SECOND PASS
  dim3 singleGrid(1, 1, 1);

  int secondLoadIdx = (threadIdx.x + 1) * blockDim.x * 2 - 1;
  int secondLoadStride = 2 * blockDim.x;

  singlePassScan<<<singleGrid, dimBlock>>>(deviceOutput, deviceOutput, numElements, secondLoadIdx, secondLoadStride);
}



int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numElements; // number of elements in the list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Clearing output memory.");
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbTime_stop(GPU, "Clearing output memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float), hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 dimGrid(ceil(numElements/float(BLOCK_SIZE * 2)), 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce

  firstPassScan<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, numElements);
  secondPassScan<<<(1,1,1), dimBlock>>>(deviceInput, deviceOutput, numElements);
  add<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, deviceOutput, numElements);


  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float), hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}

