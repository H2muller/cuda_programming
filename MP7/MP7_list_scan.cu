#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from here


__global__ void singlePassScan(float *input, float *output, int len, int loadIdx, int loadStride) {
  
  __shared__ float sharedArray[BLOCK_SIZE * 2];

  if(loadIdx < len){
        sharedArray[threadIdx.x] = input[loadIdx];
  } else {
        sharedArray[threadIdx.x] = 0; 
  }

  if(loadIdx + loadStride < len){
        sharedArray[threadIdx.x + blockDim.x] = input[loadIdx + loadStride];
  } else {
        sharedArray[threadIdx.x + blockDim.x] = 0;
  }
  __syncthreads();

  
  // reduction phase
  for (int stride = 1; stride <= blockDim.x; stride *= 2) {
    __syncthreads();
    int localIdx = (threadIdx.x + 1) * stride * 2 - 1;
    if (localIdx < 2 * blockDim.x) {
      sharedArray[localIdx] += sharedArray[localIdx - stride];
    }
  }

  // reverse phase
  for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
    __syncthreads();
    int localIdx = (threadIdx.x + 1) * stride * 2 - 1;
    if (localIdx + stride < 2 * blockDim.x) {
      sharedArray[localIdx + stride] += sharedArray[localIdx];
    }
  }

  // store partial results to output
  __syncthreads();
  if (2 * blockIdx.x * blockDim.x + threadIdx.x < len) {
    output[2 * blockIdx.x * blockDim.x + threadIdx.x] = sharedArray[threadIdx.x];
  }
  if (2 * blockIdx.x * blockDim.x + threadIdx.x + blockDim.x < len) {
    output[2 * blockIdx.x * blockDim.x + threadIdx.x + blockDim.x] = sharedArray[threadIdx.x + blockDim.x];
  }
}

__global__ void scanSum(float *input, float *output, float *sum, int len, int index) {
  __shared__ float increment;

  if (threadIdx.x == 0) {
    if (blockIdx.x == 0) {
      increment = 0;
    } else {
      increment = sum[blockIdx.x - 1];
    }
  }
  __syncthreads();

  if (index < len) {
    output[index] = input[index] + increment;
  }
  if (index + blockDim.x) {
    output[index + blockDim.x] = input[index + blockDim.x] + increment;
  }
}


void recursiveScan (float *input, float *scan_buffer, float *scan_sums, float *output, int len){
  
  // FIRST PASS

  //@@ Initialize the grid and block dimensions here
  dim3 dimGrid(ceil(len/float(BLOCK_SIZE * 2)), 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);
  dim3 singleGrid(1, 1, 1);

  int firstLoadIdx = 2 * blockIdx.x * blockDim.x + threadIdx.x;
  int firstLoadStride = blockDim.x;
  singlePassScan<<<dimGrid, dimBlock>>>(input, scan_buffer, len, firstLoadIdx, firstLoadStride);

  // SECOND PASS
  int secondLoadIdx = (threadIdx.x + 1) * blockDim.x * 2 - 1;
  int secondLoadStride = 2 * blockDim.x;
  singlePassScan<<<singleGrid, dimBlock>>>(scan_buffer, scan_sums, len, secondLoadIdx, secondLoadStride);

  // SUM
  scanSum<<<dimGrid, dimBlock>>>(scan_buffer, output, scan_sums, len, firstLoadIdx);
}



int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceScanBuffer;
  float *deviceScanSums;
  float *deviceOutput;
  int numElements; // number of elements in the list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceScanBuffer, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceScanSums, 2 * BLOCK_SIZE * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Clearing output memory.");
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbTime_stop(GPU, "Clearing output memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float), hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce

  recursiveScan<<<dimGrid, dimBlock>>>(deviceInput, deviceScanBuffer, deviceScanSums, deviceOutput, numElements);


  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float), hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceScanBuffer);
  hipFree(deviceScanSums);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}

